#include <math.h>
#include <sys/time.h>
#include <time.h>

#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define CHECK_CUDA(call)                                                        \
    do {                                                                        \
        hipError_t status_ = call;                                             \
        if (status_ != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__,  \
                    hipGetErrorName(status_), hipGetErrorString(status_));    \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)

#define CHECK_CUBLAS(call)                                                      \
    do {                                                                        \
        hipblasStatus_t err = call;                                              \
        if (err != HIPBLAS_STATUS_SUCCESS) {                                     \
            fprintf(stderr, "cuBLAS error in file '%s' at line %d: %s\n",       \
                    __FILE__, __LINE__, cublasGetErrorString(err));             \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)

// Helper function to get the error string
const char* cublasGetErrorString(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
        default:
            return "Unknown cuBLAS error";
    }
}

#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define DIVUP(x, y) ((x + y - 1) / y)
#define BLOCK_SIZE 32
#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

double get_time() {
    struct timeval tv;
    gettimeofday(&tv, 0);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

__device__ __forceinline__ float warpReduceSum(float val) {
#pragma unroll
    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
        val += __shfl_xor_sync(FULL_MASK, val, offset);
    return val;
}


__global__ void mat_vec_kernel(float *C, const float *__restrict__ B, const float *__restrict__ A,
                               int n, int d, int numSerialLoads) {
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index >= d)
        return;

    A += index * n;
    B += blockIdx.y * n;
    C += blockIdx.y * d;

    float sum = 0;
    float4 w;
    float4 inp;

#pragma unroll
    for (int i = 0; i < numSerialLoads; i++) {
        int j = (i * WARP_SIZE + threadIdx.x) * 4;
        if (j < n) {
            w = *((float4 *)(&A[j]));
            inp = *((float4 *)(&B[j]));
            sum += w.x * inp.x + w.y * inp.y + w.z * inp.z + w.w * inp.w;
        }
    }

    sum = warpReduceSum(sum);
    if (threadIdx.x == 0)
        C[index] = sum;
}

void matmul_gpu_v1(float *xout, float *x, float *w, int n, int d, int batch_size) {
    int serialElements = DIVUP(n, WARP_SIZE);
    int serialLoads = DIVUP(serialElements, 4);
    dim3 blockDim(WARP_SIZE, 4);
    dim3 gridDim(DIVUP(d, 4), batch_size);
    mat_vec_kernel<<<gridDim, blockDim>>>(xout, x, w, n, d, serialLoads);
    CHECK_CUDA(hipGetLastError());
}

void matmul_gpu(float *xout, float *x, float *w, int n, int d, int batch_size) {
    float *xoutGPU, *xGPU, *wGPU;
    CHECK_CUDA(hipMalloc(&xoutGPU, sizeof(float) * d * batch_size));
    CHECK_CUDA(hipMalloc(&xGPU, sizeof(float) * n * batch_size));
    CHECK_CUDA(hipMalloc(&wGPU, sizeof(float) * d * n));
    CHECK_CUDA(hipMemcpy(xGPU, x, sizeof(float) * n * batch_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(wGPU, w, sizeof(float) * d * n, hipMemcpyHostToDevice));
    
    // WARM UP
    for (int i=0; i<= 3; ++i) {
        matmul_gpu_v1(xoutGPU, xGPU, wGPU, n, d, batch_size);
    }
    
    float elapsed_time = 0.0;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < 10; ++i) {
        matmul_gpu_v1(xoutGPU, xGPU, wGPU, n, d, batch_size);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop)); 
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("%f sec, GFLOPS: %f \n", elapsed_time, 2ll * n * d * batch_size * 10 / (elapsed_time * 1e6));

    CHECK_CUDA(hipMemcpy(xout, xoutGPU, sizeof(float) * d * batch_size, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(xoutGPU));
    CHECK_CUDA(hipFree(xGPU));
    CHECK_CUDA(hipFree(wGPU));
}

void matmul_cublas_sgemv(float *xout, float *x, float *w, int n, int d, int batch_size) {
    // cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *xGPU, *wGPU, *xoutGPU;
    CHECK_CUDA(hipMalloc(&xoutGPU, sizeof(float) * d * batch_size));
    CHECK_CUDA(hipMalloc(&xGPU, sizeof(float) * n * batch_size));
    CHECK_CUDA(hipMalloc(&wGPU, sizeof(float) * d * n));
    CHECK_CUDA(hipMemcpy(xGPU, x, sizeof(float) * n * batch_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(wGPU, w, sizeof(float) * d * n, hipMemcpyHostToDevice));

    // Warm-up
    float alpha = 1.0f;
    float beta = 0.0f;
    for (int i = 0; i < 3; ++i) {
        CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_T, n, d, &alpha, wGPU, n, xGPU, 1, &beta, xoutGPU, 1));
    }

    // Measure performance
    float elapsed_time = 0.0;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < 10; ++i) {
        CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_T, n, d, &alpha, wGPU, n, xGPU, 1, &beta, xoutGPU, 1));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

    // Output performance
    printf("%f sec, GFLOPS: %f \n", elapsed_time, 2ll * n * d * batch_size * 10 / (elapsed_time * 1e6));

    CHECK_CUDA(hipMemcpy(xout, xoutGPU, sizeof(float) * d * batch_size, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(xoutGPU));
    CHECK_CUDA(hipFree(xGPU));
    CHECK_CUDA(hipFree(wGPU));
    CHECK_CUBLAS(hipblasDestroy(handle));
}

void matmul_cpu(float *xout, float *x, float *w, int n, int d, int batch_size) {
    for (int req_id = 0; req_id < batch_size; req_id++) {
        for (int i = 0; i < d; i++) {
            float val = 0.0f;
            for (int j = 0; j < n; j++) {
                val += w[i * n + j] * x[req_id * n + j];
            }
            xout[req_id * d + i] = val;
        }
    }
}

void vec_diff_check(float *original, float *changed, int size, bool verbose) {
    float max_diff = 0.0F;
    double total_diff = 0.0F;
    if (verbose) printf("size: %d\n", size);
    for (int i = 0; i < size; ++i) {
        if (i > size - 10 && verbose)
            printf("%d CPU: %f, GPU: %f\n", i, original[i], changed[i]);
        float diff = fabsf(original[i] - changed[i]);
        max_diff = fmaxf(max_diff, diff);
        total_diff += diff;
    }
    printf("Comparing CPU and GPU: Max diff %f, Avg diff: %f\n", max_diff, total_diff / size);
}

float *alloc_mat(int R, int C) {
    float *m;
    CHECK_CUDA(hipHostMalloc(&m, sizeof(float) * R * C, hipHostMallocDefault));
    return m;
}

void rand_mat(float *m, int R, int C) {
    for (int i = 0; i < R; i++) {
        for (int j = 0; j < C; j++) {
            m[i * C + j] = (float)rand() / (float)RAND_MAX - 0.5;
        }
    }
}

void print_mat(float *m, int R, int C) {
    for (int i = 0; i < MIN(R, 10); ++i) {
        for (int j = 0; j < MIN(C, 10); ++j) {
            printf("%+.3f ", m[i * C + j]);
        }
        printf("\n");
    }
}

void test_matmul(bool verbose) {
    int n, d;
    n = 4096;
    d = 11008;
    int batch_size = 1;
    printf("Matmul validating with n = %d, d = %d, batch_size = %d...\n", n, d, batch_size);

    float *A = alloc_mat(d, n);
    float *B = alloc_mat(batch_size, n);
    float *C_cpu = alloc_mat(batch_size, d);
    float *C_gpu = alloc_mat(batch_size, d);
    rand_mat(A, d, n);
    rand_mat(B, batch_size, n);

    matmul_cublas_sgemv(C_gpu, B, A, n, d, batch_size);
    matmul_gpu(C_gpu, B, A, n, d, batch_size);

    // matmul_cpu(C_cpu, B, A, n, d, batch_size);

    // vec_diff_check(C_cpu, C_gpu, d * batch_size, verbose);

    // printf("A: \n");
    // print_mat(A, d, n);
    // printf("\n");

    // printf("B: \n");
    // print_mat(B, batch_size, n);
    // printf("\n");

    // printf("C_gpu: \n");
    // print_mat(C_gpu, batch_size, d);
    // printf("\n");

    // printf("C_cpu: \n");
    // print_mat(C_cpu, batch_size, d);
    // printf("\n");
}

int main(int argc, char **argv) {
    // Seed the random number generator with the current time
    srand(time(NULL));
    bool verbose = atoi(argv[0]);
    test_matmul(verbose);
}